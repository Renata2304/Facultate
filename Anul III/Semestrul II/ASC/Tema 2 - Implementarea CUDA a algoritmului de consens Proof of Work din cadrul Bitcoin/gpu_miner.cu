#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include "../include/utils.cuh"
#include <string.h>
#include <stdlib.h>
#include <inttypes.h>

#define GL_BLOCKS_NO (1 << 8)         // Number of blocks in the grid
#define GL_BLOCK_SIZE (1 << 9)        // Size of each block
#define GL_CORES_NO (1 << 17)         // Number of cores

__global__ void findNonce(BYTE* block_content, size_t current_length, BYTE* block_hash, uint64_t* nonce, BYTE* difficulty, bool *found_nonce) {
	// Thread index calculation
    unsigned int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
	// Nonce range for current thread
    unsigned int start = thread_id * (double)MAX_NONCE / GL_CORES_NO;
    unsigned int end = min((thread_id + 1) * (double)MAX_NONCE / GL_CORES_NO, MAX_NONCE);

    // Local variables
    BYTE device_block_hash[SHA256_HASH_SIZE];
    BYTE local_content[BLOCK_SIZE];

    // Starting content block
    d_strcpy((char*)local_content, (const char*)block_content);

	// Search for a valid nonce in the assigned range
    for (unsigned int crt_nonce = start; crt_nonce < end; crt_nonce++) {
		if (*found_nonce)
        	break;

    	char nonce_string[NONCE_SIZE];
        // Convert nonce to string
        intToString(crt_nonce, nonce_string);

        // Concatenate current nonce
        d_strcpy((char*)local_content + current_length, (const char*)nonce_string);

        // Apply hash
        apply_sha256(local_content, d_strlen((const char*)local_content), device_block_hash, 1);

        if (compare_hashes(device_block_hash, difficulty) <= 0) {
            // Store the found nonce
            *nonce = crt_nonce;

            // Store block_hash
            d_strcpy((char*)block_hash, (const char*)device_block_hash);

			// Update found_nonce flag
            *found_nonce = true;

			// Exit the loop if a valid nonce is found
            break; 
        }
    }
}

int main(int argc, char **argv) {
	BYTE hashed_tx1[SHA256_HASH_SIZE], hashed_tx2[SHA256_HASH_SIZE], hashed_tx3[SHA256_HASH_SIZE], hashed_tx4[SHA256_HASH_SIZE],
			tx12[SHA256_HASH_SIZE * 2], tx34[SHA256_HASH_SIZE * 2], hashed_tx12[SHA256_HASH_SIZE], hashed_tx34[SHA256_HASH_SIZE],
			tx1234[SHA256_HASH_SIZE * 2], top_hash[SHA256_HASH_SIZE], block_content[BLOCK_SIZE];
	BYTE block_hash[SHA256_HASH_SIZE] = "0000000000000000000000000000000000000000000000000000000000000000"; // TODO: Update
	uint64_t nonce = 0; // TODO: Update
	size_t current_length;

	// Top hash
	apply_sha256(tx1, strlen((const char*)tx1), hashed_tx1, 1);
	apply_sha256(tx2, strlen((const char*)tx2), hashed_tx2, 1);
	apply_sha256(tx3, strlen((const char*)tx3), hashed_tx3, 1);
	apply_sha256(tx4, strlen((const char*)tx4), hashed_tx4, 1);
	strcpy((char *)tx12, (const char *)hashed_tx1);
	strcat((char *)tx12, (const char *)hashed_tx2);
	apply_sha256(tx12, strlen((const char*)tx12), hashed_tx12, 1);
	strcpy((char *)tx34, (const char *)hashed_tx3);
	strcat((char *)tx34, (const char *)hashed_tx4);
	apply_sha256(tx34, strlen((const char*)tx34), hashed_tx34, 1);
	strcpy((char *)tx1234, (const char *)hashed_tx12);
	strcat((char *)tx1234, (const char *)hashed_tx34);
	apply_sha256(tx1234, strlen((const char*)tx34), top_hash, 1);

	// prev_block_hash + top_hash
	strcpy((char*)block_content, (const char*)prev_block_hash);
	strcat((char*)block_content, (const char*)top_hash);
	current_length = strlen((char*) block_content);

	// Allocate memory and copy data on device
	BYTE *device_block_content;
	hipMalloc(&device_block_content, sizeof(BYTE) * BLOCK_SIZE);
    hipMemcpy(device_block_content, block_content, sizeof(BYTE) * BLOCK_SIZE, hipMemcpyHostToDevice);

	BYTE *device_block_hash;
    hipMalloc(&device_block_hash, sizeof(BYTE) * SHA256_HASH_SIZE);
    hipMemcpy(device_block_hash, block_hash, sizeof(BYTE) * SHA256_HASH_SIZE, hipMemcpyHostToDevice);

    uint64_t *device_nonce;
    hipMalloc(&device_nonce, sizeof(uint64_t));
    hipMemcpy(device_nonce, &nonce, sizeof(uint64_t), hipMemcpyHostToDevice);

	BYTE *device_difficulty;
	hipMalloc(&device_difficulty, sizeof(BYTE) * SHA256_HASH_SIZE);
	hipMemcpy(device_difficulty, difficulty_5_zeros, sizeof(BYTE) * SHA256_HASH_SIZE, hipMemcpyHostToDevice);

	// Initialize device found_nonce flag
	bool found_nonce = false;
	bool *device_found_nonce;
    hipMalloc(&device_found_nonce, sizeof(bool));
    hipMemcpy(device_found_nonce, &found_nonce, sizeof(bool), hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	startTiming(&start, &stop);

	findNonce<<<GL_BLOCKS_NO, GL_BLOCK_SIZE>>>(device_block_content, current_length, device_block_hash, device_nonce, device_difficulty, device_found_nonce);

	float seconds = stopTiming(&start, &stop);

	// Copy results back from device
	hipMemcpy(&nonce, device_nonce, sizeof(uint64_t), hipMemcpyDeviceToHost);
    hipMemcpy(block_hash, device_block_hash, sizeof(BYTE) * SHA256_HASH_SIZE, hipMemcpyDeviceToHost);

	printResult(block_hash, nonce, seconds);

	// Cleanup
    hipFree(device_block_content);
    hipFree(device_block_hash);
	hipFree(device_difficulty);
    hipFree(device_nonce);
	hipFree(device_found_nonce);

	return 0;
}
